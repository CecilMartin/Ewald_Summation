#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>
#include <ctime>
#include <fstream>
#include <iostream>
#include <complex>
#include <hipfft/hipfft.h>
#include <cstdio>
#include <stdlib.h>
#include <omp.h>
#include <chrono>
#include <iomanip>
#include "fftw3.h"
#include "ewald.h"
#include "utils.h"

using namespace std;
using namespace chrono;

void initialize(){
  srand((unsigned int) 100005);
  nx = 64;
  ny = 64;
  nz = 64;
  Lx = 2.0;
  Ly = 2.0;
  Lz = 2.0;
  eta = 1.15;
  xi = 12;
  np = 2048;
  px = 24;
  py = 24;
  pz = 24;
  repeat_x = 2;
  repeat_y = 2;
  repeat_z = 2;
  grid = (double*)calloc(np*DIM,sizeof(double));
  particle = (double*)calloc(np*DIM,sizeof(double));
  strength = (double*)calloc(np*DIM,sizeof(double));
  vel = (double*)calloc(np*DIM,sizeof(double));
  for (int i = 0; i<np; i++){
    particle[DIM*i+0] = rand()*1.0/RAND_MAX*Lx;
    particle[DIM*i+1] = rand()*1.0/RAND_MAX*Ly;
    particle[DIM*i+2] = rand()*1.0/RAND_MAX*Lz;
    strength[DIM*i+0] = 1.0;
    strength[DIM*i+1] = 1.0;
    strength[DIM*i+2] = 1.0;
  }
  outputfile = "../results/vel2.txt";
  cout << "System initialized! # of particles: " << np << '\n';
}


void initialize_readinput(int N, int num_p, int P, double eta_in,
  int rp, double L, double xi_in){
  srand((unsigned int) 100005);
  nx = N;
  ny = N;
  nz = N;
  Lx = L;
  Ly = L;
  Lz = L;
  eta = eta_in;
  xi = xi_in;
  np = num_p;
  px = P;
  py = P;
  pz = P;
  repeat_x = rp;
  repeat_y = rp;
  repeat_z = rp;
  grid = (double*)calloc(np*DIM,sizeof(double));
  particle = (double*)calloc(np*DIM,sizeof(double));
  strength = (double*)calloc(np*DIM,sizeof(double));
  vel = (double*)calloc(np*DIM,sizeof(double));
  for (int i = 0; i<np; i++){
    particle[DIM*i+0] = rand()*1.0/RAND_MAX*Lx;
    particle[DIM*i+1] = rand()*1.0/RAND_MAX*Ly;
    particle[DIM*i+2] = rand()*1.0/RAND_MAX*Lz;
    strength[DIM*i+0] = 1.0;
    strength[DIM*i+1] = 1.0;
    strength[DIM*i+2] = 1.0;
  }
  outputfile = "../results/vel2.txt";
  cout << "System initialized! # of particles: " << np << '\n';
}

void realfunc(double x, double y, double z, double xi, double *st1, double *st2, double *v){
  double r2 = x*x + y*y + z*z;
  double r = sqrt(r2);
  double e1 = exp(-xi*xi*r2);
  double tmp1[3], tmp2[3];
  double coef[2] = {2*(xi*e1/sqrt(M_PI)/r2+erfc(xi*r)/2/r2/r),4*xi/sqrt(M_PI)*e1};
  double *st = st1;
  tmp1[0] = (r2+x*x)*st[0]+(x*y)*st[1]+(x*z)*st[2];
  tmp1[1] = (y*x)*st[0]+(r2+y*y)*st[1]+(y*z)*st[2];
  tmp1[2] = (z*x)*st[0]+(z*y)*st[1]+(r2+z*z)*st[2];
  st = st2;
  tmp2[0] = (r2+x*x)*st[0]+(x*y)*st[1]+(x*z)*st[2];
  tmp2[1] = (y*x)*st[0]+(r2+y*y)*st[1]+(y*z)*st[2];
  tmp2[2] = (z*x)*st[0]+(z*y)*st[1]+(r2+z*z)*st[2];
  v[0] = coef[0]*tmp1[0]-coef[1]*st1[0];
  v[1] = coef[0]*tmp1[1]-coef[1]*st1[1];
  v[2] = coef[0]*tmp1[2]-coef[1]*st1[2];
  v[3] = coef[0]*tmp2[0]-coef[1]*st2[0];
  v[4] = coef[0]*tmp2[1]-coef[1]*st2[1];
  v[5] = coef[0]*tmp2[2]-coef[1]*st2[2];
  return;
}

void realfuncOMP(double x, double y, double z, double xi, double *st, double *v){
  double r2 = x*x + y*y + z*z;
  double r = sqrt(r2);
  double e1 = exp(-xi*xi*r2);
  double tmp1[3];
  double coef[2] = {2*(xi*e1/sqrt(M_PI)/r2+erfc(xi*r)/2/r2/r),4*xi/sqrt(M_PI)*e1};
  tmp1[0] = (r2+x*x)*st[0]+(x*y)*st[1]+(x*z)*st[2];
  tmp1[1] = (y*x)*st[0]+(r2+y*y)*st[1]+(y*z)*st[2];
  tmp1[2] = (z*x)*st[0]+(z*y)*st[1]+(r2+z*z)*st[2];
  v[0] = coef[0]*tmp1[0]-coef[1]*st[0];
  v[1] = coef[0]*tmp1[1]-coef[1]*st[1];
  v[2] = coef[0]*tmp1[2]-coef[1]*st[2];
  return;
}


void realspace(){
  double rx, ry, rz;
  double v[6];
  Timer tt;
  tt.tic();
  for (int i = 0; i < np; i++){
    for (int j = i+1; j < np; j++){
      for (int px = -repeat_x; px < repeat_x; px++){
        for (int py = -repeat_y; py < repeat_y; py++){
          for (int pz = -repeat_z; pz < repeat_z; pz++){
            if (px == 0 && py == 0 && pz == 0){
              if (i != j) {
                rx = particle[DIM*j+0]-particle[DIM*i+0];
                ry = particle[DIM*j+1]-particle[DIM*i+1];
                rz = particle[DIM*j+2]-particle[DIM*i+2];
                realfunc(rx, ry, rz, xi, &(strength[DIM*i]), &(strength[DIM*j]), v);
                vel[DIM*i+0] += v[3];
                vel[DIM*i+1] += v[4];
                vel[DIM*i+2] += v[5];

                vel[DIM*j+0] += v[0];
                vel[DIM*j+1] += v[1];
                vel[DIM*j+2] += v[2];
              }
            }
            else{
              rx = particle[DIM*j+0]+Lx*px-particle[DIM*i+0];
              ry = particle[DIM*j+1]+Ly*py-particle[DIM*i+1];
              rz = particle[DIM*j+2]+Lz*pz-particle[DIM*i+2];
              realfunc(rx, ry, rz, xi, &(strength[DIM*i]), &(strength[DIM*j]), v);
              vel[DIM*i+0] += v[3];
              vel[DIM*i+1] += v[4];
              vel[DIM*i+2] += v[5];

              vel[DIM*j+0] += v[0];
              vel[DIM*j+1] += v[1];
              vel[DIM*j+2] += v[2];
            }
          }
        }
      }
    }
  }
  printf("Real space part finished with %f(s)\n",tt.toc());
}


void realspaceOMP(int num_threads){
  #if defined(_OPENMP)
	int threads_all = omp_get_num_procs();
	cout << "Number of cpus in this machine: " << threads_all << endl;
	omp_set_num_threads(num_threads);
	cout << "Use " << num_threads << " threads" << endl;
	#endif
  double rx, ry, rz;
  double tt = omp_get_wtime();
  #pragma omp parallel private(rx,ry,rz)
  {
  #pragma omp for
  for (int i = 0; i < np; i++){
    // printf("i=%d, from thread = %d\n", i, omp_get_thread_num());
    double v[3];
    for (int j = 0; j < np; j++){
      for (int px = -repeat_x; px < repeat_x; px++){
        for (int py = -repeat_y; py < repeat_y; py++){
          for (int pz = -repeat_z; pz < repeat_z; pz++){
            if (px == 0 && py == 0 && pz == 0){
              if (i != j) {
                rx = particle[DIM*j+0]-particle[DIM*i+0];
                ry = particle[DIM*j+1]-particle[DIM*i+1];
                rz = particle[DIM*j+2]-particle[DIM*i+2];
                realfuncOMP(rx, ry, rz, xi, &(strength[DIM*j]), v);
                vel[DIM*i+0] += v[0];
                vel[DIM*i+1] += v[1];
                vel[DIM*i+2] += v[2];
              }
            }
            else{
              rx = particle[DIM*j+0]+Lx*px-particle[DIM*i+0];
              ry = particle[DIM*j+1]+Ly*py-particle[DIM*i+1];
              rz = particle[DIM*j+2]+Lz*pz-particle[DIM*i+2];
              realfuncOMP(rx, ry, rz, xi, &(strength[DIM*j]), v);
              vel[DIM*i+0] += v[0];
              vel[DIM*i+1] += v[1];
              vel[DIM*i+2] += v[2];
            }
          }
        }
      }
    }
  }
  }
  #pragma omp barrier
  printf("OpenMP!! \nReal space part finished with %f(s)\n",(omp_get_wtime()-tt));
}


void Gaussian_Gridding_type1_OMP(double *H){
  double tt = omp_get_wtime();
  double hx = Lx / nx, hy = Ly / ny, hz = Lz / nz;
  double hx_sq = hx * hx, hy_sq = hy * hy, hz_sq = hz * hz;
  int ig, jg, kg;
  double a =  2 * xi * xi / eta;
  double xp, yp, zp, xp_o, yp_o, zp_o;
  int ip, jp, kp;
  double* E3_x = (double*) calloc(px+1, sizeof(double));
  double* E3_y = (double*) calloc(py+1, sizeof(double));
  double* E3_z = (double*) calloc(pz+1, sizeof(double));
  double E1_x, E1_y, E1_z, E2_x, E2_y, E2_z;
  double* E2_xl = (double*) calloc(2*px, sizeof(double));
  double* E2_yl = (double*) calloc(2*py, sizeof(double));
  double* E2_zl = (double*) calloc(2*pz, sizeof(double));
  double V0, Vx, Vy, Vz;
  for (long i = 0; i <= px; i++) {
    E3_x[i] = exp(-a*i*i*hx_sq);
  }
  for (long i = 0; i <= py; i++) {
    E3_y[i] = exp(-a*i*i*hy_sq);
  }
  for (long i = 0; i <= pz; i++) {
    E3_z[i] = exp(-a*i*i*hz_sq);
  }
  for (long n = 0; n < np; n++){
    xp = particle[DIM*n+0];
    yp = particle[DIM*n+1];
    zp = particle[DIM*n+2];
    ip = xp/hx; jp = yp/hy; kp = zp/hz;
    xp_o = xp - ip*hx; yp_o = yp - jp*hy; zp_o = zp - kp*hz;

    E1_x = exp(-a*xp_o*xp_o);
    E1_y = exp(-a*yp_o*yp_o);
    E1_z = exp(-a*zp_o*zp_o);
    E2_x = exp(2*a*xp_o*hx);
    E2_y = exp(2*a*yp_o*hy);
    E2_z = exp(2*a*zp_o*hz);
    for (long i = - px+1; i <= px; i++) {
      E2_xl[i+px-1] = pow(E2_x, i);
    }
    for (long j = -py+1; j <= py; j++) {
      E2_yl[j+py-1] = pow(E2_y, j);
    }
    for (long k = -pz+1; k <= pz; k++) {
      E2_zl[k+pz-1] = pow(E2_z, k);
    }
    V0 = E1_x * E1_y * E1_z;
    #pragma omp parallel for schedule(dynamic, 1) private(Vx, Vy, Vz, ig, jg, kg)
    for (long i = - px+1; i <= px; i++){
      // printf("i=%d, from thread = %d\n", i, omp_get_thread_num());
      Vx = V0 * E2_xl[i+px-1] * E3_x[abs(i)];
      for (long j = -py+1; j <= py; j++){
        Vy = Vx * E2_yl[j+py-1] * E3_y[abs(j)];
        for (long k = -pz+1; k <= pz; k++){
          Vz = Vy * E2_zl[k+pz-1] * E3_z[abs(k)];
          ig = (ip+i+nx) % nx; jg = (jp+j+ny) % ny; kg = (kp+k+nz) % nz;
          for (long m = 0; m < DIM; m++){
            H[kg + nz*(jg + ny*(ig + m*nx))] += Vz * strength[DIM*n+m];
          }
        }
      }
    }
    #pragma omp barrier
  }
  printf("OpenMP Gaussian_Gridding_type1 finished with %f(s)\n",(omp_get_wtime()-tt));
  free(E3_x);free(E3_y);free(E3_z);
  free(E2_xl);free(E2_yl);free(E2_zl);
  return;
}

void Gaussian_Gridding_type1(double *H){
  double tt = omp_get_wtime();
  double hx = Lx / nx, hy = Ly / ny, hz = Lz / nz;
  double hx_sq = hx * hx, hy_sq = hy * hy, hz_sq = hz * hz;
  int ig, jg, kg;
  double a =  2 * xi * xi / eta;
  double xp, yp, zp, xp_o, yp_o, zp_o;
  int ip, jp, kp;
  double* E3_x = (double*) calloc(px+1, sizeof(double));
  double* E3_y = (double*) calloc(py+1, sizeof(double));
  double* E3_z = (double*) calloc(pz+1, sizeof(double));
  double E1_x, E1_y, E1_z, E2_x, E2_y, E2_z;
  double* E2_xl = (double*) calloc(2*px, sizeof(double));
  double* E2_yl = (double*) calloc(2*py, sizeof(double));
  double* E2_zl = (double*) calloc(2*pz, sizeof(double));
  double V0, Vx, Vy, Vz;
  for (long i = 0; i <= px; i++) {
    E3_x[i] = exp(-a*i*i*hx_sq);
  }
  for (long i = 0; i <= py; i++) {
    E3_y[i] = exp(-a*i*i*hy_sq);
  }
  for (long i = 0; i <= pz; i++) {
    E3_z[i] = exp(-a*i*i*hz_sq);
  }
  for (long n = 0; n < np; n++){
    xp = particle[DIM*n+0];
    yp = particle[DIM*n+1];
    zp = particle[DIM*n+2];
    ip = xp/hx; jp = yp/hy; kp = zp/hz;
    xp_o = xp - ip*hx; yp_o = yp - jp*hy; zp_o = zp - kp*hz;
    E1_x = exp(-a*xp_o*xp_o);
    E1_y = exp(-a*yp_o*yp_o);
    E1_z = exp(-a*zp_o*zp_o);
    E2_x = exp(2*a*xp_o*hx);
    E2_y = exp(2*a*yp_o*hy);
    E2_z = exp(2*a*zp_o*hz);
    for (long i = - px+1; i <= px; i++) {
      E2_xl[i+px-1] = pow(E2_x, i);
    }
    for (long j = -py+1; j <= py; j++) {
      E2_yl[j+py-1] = pow(E2_y, j);
    }
    for (long k = -pz+1; k <= pz; k++) {
      E2_zl[k+pz-1] = pow(E2_z, k);
    }
    V0 = E1_x * E1_y * E1_z;
    for (long i = - px+1; i <= px; i++){
      // printf("i=%d, from thread = %d\n", i, omp_get_thread_num());
      Vx = V0 * E2_xl[i+px-1] * E3_x[abs(i)];
      for (long j = -py+1; j <= py; j++){
        Vy = Vx * E2_yl[j+py-1] * E3_y[abs(j)];
        for (long k = -pz+1; k <= pz; k++){
          Vz = Vy * E2_zl[k+pz-1] * E3_z[abs(k)];
          ig = (ip+i+nx) % nx; jg = (jp+j+ny) % ny; kg = (kp+k+nz) % nz;
          for (long m = 0; m < DIM; m++){
            H[kg + nz*(jg + ny*(ig + m*nx))] += Vz * strength[DIM*n+m];
          }
        }
      }
    }
  }
  printf("Gaussian_Gridding_type1 finished with %f(s)\n",(omp_get_wtime()-tt));
  free(E3_x);free(E3_y);free(E3_z);
  free(E2_xl);free(E2_yl);free(E2_zl);
  return;
}

void Gaussian_Gridding_type2_OMP(double* H){
  double tt = omp_get_wtime();
  double hx = Lx / nx, hy = Ly / ny, hz = Lz / nz;
  double scale_factor = hx*hy*hz * pow(2*xi*xi/(M_PI*eta), 1.5);
  double hx_sq = hx * hx, hy_sq = hy * hy, hz_sq = hz * hz;
  int ig, jg, kg;
  double a = 2 * xi * xi / eta;
  double xp, yp, zp, xp_o, yp_o, zp_o;
  int ip, jp, kp;
  double* E3_x = (double*) calloc(px+1, sizeof(double));
  double* E3_y = (double*) calloc(py+1, sizeof(double));
  double* E3_z = (double*) calloc(pz+1, sizeof(double));
  double E1_x, E1_y, E1_z, E2_x, E2_y, E2_z;
  double V0, Vx, Vy, Vz;
  for (long i = 0; i <= px; i++) {
    E3_x[i] = exp(-a*i*i*hx_sq);
  }
  for (long i = 0; i <= py; i++) {
    E3_y[i] = exp(-a*i*i*hy_sq);
  }
  for (long i = 0; i <= pz; i++) {
    E3_z[i] = exp(-a*i*i*hz_sq);
  }
  #pragma omp parallel for private(xp, yp, zp, ip, jp, kp, xp_o, yp_o, zp_o, E1_x, E1_y, E1_z, E2_x, E2_y, E2_z,V0, Vx, Vy, Vz, ig, jg, kg)
  for (long n = 0; n < np; n++){
    double* E2_xl = (double*) calloc(2*px, sizeof(double));
    double* E2_yl = (double*) calloc(2*py, sizeof(double));
    double* E2_zl = (double*) calloc(2*pz, sizeof(double));
    xp = particle[DIM*n+0];
    yp = particle[DIM*n+1];
    zp = particle[DIM*n+2];
    ip = xp/hx; jp = yp/hy; kp = zp/hz;
    xp_o = xp - ip*hx; yp_o = yp - jp*hy; zp_o = zp - kp*hz;
    E1_x = exp(-a*xp_o*xp_o);
    E1_y = exp(-a*yp_o*yp_o);
    E1_z = exp(-a*zp_o*zp_o);
    E2_x = exp(2*a*xp_o*hx);
    E2_y = exp(2*a*yp_o*hy);
    E2_z = exp(2*a*zp_o*hz);
    for (long i = - px+1; i <= px; i++) {
      E2_xl[i+px-1] = pow(E2_x, i);
    }
    for (long j = -py+1; j <= py; j++) {
      E2_yl[j+py-1] = pow(E2_y, j);
    }
    for (long k = -pz+1; k <= pz; k++) {
      E2_zl[k+pz-1] = pow(E2_z, k);
    }
    V0 = E1_x * E1_y * E1_z;
    for (long i = - px+1; i <= px; i++){
      Vx = V0 * E2_xl[i+px-1] * E3_x[abs(i)];
      for (long j = -py+1; j <= py; j++){
        Vy = Vx * E2_yl[j+py-1] * E3_y[abs(j)];
        for (long k = -pz+1; k <= pz; k++){
          Vz = Vy * E2_zl[k+pz-1] * E3_z[abs(k)];
          ig = (ip+i+nx) % nx; jg = (jp+j+ny) % ny; kg = (kp+k+nz) % nz;
          for (long m = 0; m < DIM; m++){
            vel[DIM*n+m] += scale_factor * Vz * H[kg + ny*(jg + nz*(ig + m*nx))];
          }
        }
      }
    }
    free(E2_xl);free(E2_yl);free(E2_zl);
  }
  #pragma omp barrier
  printf("OpenMP Gaussian_Gridding_type2 finished with %f(s)\n",(omp_get_wtime()-tt));
  free(E3_x);free(E3_y);free(E3_z);
  return;
}

void Gaussian_Gridding_type2(double* H){
  double tt = omp_get_wtime();
  double hx = Lx / nx, hy = Ly / ny, hz = Lz / nz;
  double scale_factor = hx*hy*hz * pow(2*xi*xi/(M_PI*eta), 1.5);
  double hx_sq = hx * hx, hy_sq = hy * hy, hz_sq = hz * hz;
  int ig, jg, kg;
  double a = 2 * xi * xi / eta;
  double xp, yp, zp, xp_o, yp_o, zp_o;
  int ip, jp, kp;
  double* E3_x = (double*) calloc(px+1, sizeof(double));
  double* E3_y = (double*) calloc(py+1, sizeof(double));
  double* E3_z = (double*) calloc(pz+1, sizeof(double));
  double E1_x, E1_y, E1_z, E2_x, E2_y, E2_z;
  double* E2_xl = (double*) calloc(2*px, sizeof(double));
  double* E2_yl = (double*) calloc(2*py, sizeof(double));
  double* E2_zl = (double*) calloc(2*pz, sizeof(double));
  double V0, Vx, Vy, Vz;
  for (long i = 0; i <= px; i++) {
    E3_x[i] = exp(-a*i*i*hx_sq);
  }
  for (long i = 0; i <= py; i++) {
    E3_y[i] = exp(-a*i*i*hy_sq);
  }
  for (long i = 0; i <= pz; i++) {
    E3_z[i] = exp(-a*i*i*hz_sq);
  }
  for (long n = 0; n < np; n++){
    xp = particle[DIM*n+0];
    yp = particle[DIM*n+1];
    zp = particle[DIM*n+2];
    ip = xp/hx; jp = yp/hy; kp = zp/hz;
    xp_o = xp - ip*hx; yp_o = yp - jp*hy; zp_o = zp - kp*hz;
    E1_x = exp(-a*xp_o*xp_o);
    E1_y = exp(-a*yp_o*yp_o);
    E1_z = exp(-a*zp_o*zp_o);
    E2_x = exp(2*a*xp_o*hx);
    E2_y = exp(2*a*yp_o*hy);
    E2_z = exp(2*a*zp_o*hz);
    for (long i = - px+1; i <= px; i++) {
      E2_xl[i+px-1] = pow(E2_x, i);
    }
    for (long j = -py+1; j <= py; j++) {
      E2_yl[j+py-1] = pow(E2_y, j);
    }
    for (long k = -pz+1; k <= pz; k++) {
      E2_zl[k+pz-1] = pow(E2_z, k);
    }
    V0 = E1_x * E1_y * E1_z;
    for (long i = - px+1; i <= px; i++){
      Vx = V0 * E2_xl[i+px-1] * E3_x[abs(i)];
      for (long j = -py+1; j <= py; j++){
        Vy = Vx * E2_yl[j+py-1] * E3_y[abs(j)];
        for (long k = -pz+1; k <= pz; k++){
          Vz = Vy * E2_zl[k+pz-1] * E3_z[abs(k)];
          ig = (ip+i+nx) % nx; jg = (jp+j+ny) % ny; kg = (kp+k+nz) % nz;
          for (long m = 0; m < DIM; m++){
            vel[DIM*n+m] += scale_factor * Vz * H[kg + ny*(jg + nz*(ig + m*nx))];
          }
        }
      }
    }
  }
  printf("Gaussian_Gridding_type2 finished with %f(s)\n",(omp_get_wtime()-tt));
  free(E3_x);free(E3_y);free(E3_z);
  free(E2_xl);free(E2_yl);free(E2_zl);
  return;
}

void FFT3DGPU(double *H, complex<double> *odata){
  hipfftHandle plan;
  hipfftDoubleReal *data;
  hipfftDoubleComplex *data1;
  int n[DIM] = {nx, ny, nz};
  /* Create a 3D FFT plan. */
  if (hipfftPlanMany(&plan, DIM, n,
    NULL, 1, nx*ny*nz, // *inembed, istride, idist
    NULL, 1, nx*ny*(nz/2+1), // *onembed, ostride, odist
    HIPFFT_D2Z, 3) != HIPFFT_SUCCESS){
      fprintf(stderr, "CUFFT error: Plan creation failed");
      return;
  }
  Timer tt, tt1;
  tt.tic();
  hipMalloc((void**)&data, sizeof(hipfftDoubleReal)*nx*ny*nz*3);
  hipMalloc((void**)&data1, sizeof(hipfftDoubleComplex)*nx*ny*(nz/2+1)*3);
  tt1.tic();
  hipMemcpy(data,H,nx*ny*nz*3*sizeof(hipfftDoubleReal),hipMemcpyHostToDevice);
  printf("Memcpy from Host To Device %f(s)\n",(tt1.toc()));
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to allocate\n");
    return;
  }

  /* Use the CUFFT plan to transform the signal in place. */
  if (hipfftExecD2Z(plan, data, data1) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecD2Z Forward failed");
    return;
  }
  if (hipDeviceSynchronize() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to synchronize\n");
    return;
  }
  hipMemcpy(odata,data1,nx*ny*(nz/2+1)*3*sizeof(hipfftDoubleComplex),hipMemcpyDeviceToHost);
  hipfftDestroy(plan);
  hipFree(data);
  hipFree(data1);
  hipDeviceSynchronize();
  printf("FFT finished with %f(s)\n",(tt.toc()));
  return;
}
void IFFT3DGPU(complex<double> *H, double *odata){
  hipfftHandle plan;
  hipfftDoubleReal *data;
  hipfftDoubleComplex *data1;
  int n[DIM] = {nx, ny, nz};
  hipMalloc((void**)&data, sizeof(hipfftDoubleReal)*nx*ny*nz*3);
  hipMalloc((void**)&data1, sizeof(hipfftDoubleComplex)*nx*ny*(nz/2+1)*3);
  hipMemcpy(data1,H,nx*ny*(nz/2+1)*3*sizeof(hipfftDoubleComplex),hipMemcpyHostToDevice);
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to allocate\n");
    return;
  }
  /* Create a 3D FFT plan. */
  if (hipfftPlanMany(&plan, DIM, n,
  NULL, 1, nx*ny*(nz/2+1), // *inembed, istride, idist
  NULL, 1, nx*ny*nz, // *onembed, ostride, odist
  HIPFFT_Z2D, 3) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: Plan creation failed");
    return;
  }
  Timer tt;
  tt.tic();
  /* Use the CUFFT plan to transform the signal in place. */
  if (hipfftExecZ2D(plan, data1, data) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecZ2D Reverse failed");
    return;
  }
  if (hipDeviceSynchronize() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to synchronize\n");
    return;
  }
  hipMemcpy(odata,data,nx*ny*(nz)*3*sizeof(hipfftDoubleReal),hipMemcpyDeviceToHost);
  hipfftDestroy(plan);
  hipFree(data);
  hipFree(data1);
  for (int idim = 0; idim<3; idim++){
    for (int i = 0; i<nx; i++){
      for (int j = 0; j<ny; j++){
        for (int k = 0; k<nz; k++){
          odata[idim*(nx*ny*nz)+i*(ny*nz)+j*nz+k]/=(nx*ny*nz);
        }
      }
    }
  }
  hipDeviceSynchronize();
  printf("IFFT finished with %f(s)\n",(tt.toc()));
  return;
}

void FFT3D(double *H, complex<double> *odata){
  fftw_complex *out;
  double *in;
  fftw_plan p;
  Timer tt;
  tt.tic();
  out = (fftw_complex*) fftw_malloc(sizeof(fftw_complex) * nx*ny*(nz/2+1));
  in = (double*) fftw_malloc(sizeof(double) * nx*ny*(nz));
  p = fftw_plan_dft_r2c_3d(nx, ny, nz, in, out, FFTW_ESTIMATE);

  memcpy(in, &(H[0*nx*ny*(nz)]), sizeof(double)*nx*ny*(nz));
  fftw_execute(p); /* repeat as needed */
  memcpy(&(odata[0*nx*ny*(nz/2+1)]), out, sizeof(complex<double>)*nx*ny*(nz/2+1));

  memcpy(in, &(H[1*nx*ny*(nz)]), sizeof(double)*nx*ny*(nz));
  fftw_execute(p); /* repeat as needed */
  memcpy(&(odata[1*nx*ny*(nz/2+1)]), out, sizeof(complex<double>)*nx*ny*(nz/2+1));

  memcpy(in, &(H[2*nx*ny*(nz)]), sizeof(double)*nx*ny*(nz));
  fftw_execute(p); /* repeat as needed */
  memcpy(&(odata[2*nx*ny*(nz/2+1)]), out, sizeof(complex<double>)*nx*ny*(nz/2+1));
  printf("FFTW finished with %f(s)\n",(tt.toc()));
  fftw_destroy_plan(p);
  fftw_free(in); fftw_free(out);
  return;
}

void IFFT3D(complex<double> *H, double *odata){
  fftw_complex *in;
  double *out;
  fftw_plan p;
  Timer tt;
  tt.tic();
  in = (fftw_complex*) fftw_malloc(sizeof(fftw_complex) * nx*ny*(nz/2+1));
  out = (double*) fftw_malloc(sizeof(double) * nx*ny*(nz));
  p = fftw_plan_dft_c2r_3d(nx, ny, nz, in, out, FFTW_ESTIMATE);

  memcpy(in, &(H[0*nx*ny*(nz/2+1)]), sizeof(complex<double>)*nx*ny*(nz/2+1));
  fftw_execute(p); /* repeat as needed */
  memcpy(&(odata[0*nx*ny*(nz)]), out, sizeof(double)*nx*ny*(nz));

  memcpy(in, &(H[1*nx*ny*(nz/2+1)]), sizeof(complex<double>)*nx*ny*(nz/2+1));
  fftw_execute(p); /* repeat as needed */
  memcpy(&(odata[1*nx*ny*(nz)]), out, sizeof(double)*nx*ny*(nz));

  memcpy(in, &(H[2*nx*ny*(nz/2+1)]), sizeof(complex<double>)*nx*ny*(nz/2+1));
  fftw_execute(p); /* repeat as needed */
  memcpy(&(odata[2*nx*ny*(nz)]), out, sizeof(double)*nx*ny*(nz));

  printf("IFFTW finished with %f(s)\n",(tt.toc()));
  for (int idim = 0; idim<3; idim++){
    for (int i = 0; i<nx; i++){
      for (int j = 0; j<ny; j++){
        for (int k = 0; k<nz; k++){
          odata[idim*(nx*ny*nz)+i*(ny*nz)+j*nz+k]/=(nx*ny*nz);
        }
      }
    }
  }
  fftw_destroy_plan(p);
  fftw_free(in); fftw_free(out);
  return;
}

void kspaceParallel(){
  Timer tt;
  tt.tic();
  double *Hx;
  Hx = (double*)malloc(sizeof(double)*nx*ny*(nz)*3);
  Gaussian_Gridding_type1_OMP(Hx);
  complex<double> *Hx_hat;
  Hx_hat = (complex<double>*)malloc(sizeof(complex<double>)*nx*ny*(nz/2+1)*3);
  FFT3DGPU(Hx, Hx_hat);
  complex<double> Hx_tilde[3*(nx)*(ny)*(nz/2+1)];
  double kx, ky, kz, k2, e1;
  double kx0=2*M_PI/Lx, ky0=2*M_PI/Ly, kz0=2*M_PI/Lz;
  for (int i = 0; i<nx; i++){
    for (int j = 0; j<ny; j++){
      for (int k = 0; k<nz/2+1; k++){
        if (i==0 && j==0 && k==0){
          Hx_tilde[0*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]={0,0};
          Hx_tilde[1*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]={0,0};
          Hx_tilde[2*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]={0,0};
          continue;
        }
        if (i<nx/2){
          kx = i*kx0;
        }
        else{
          kx = (i-nx)*kx0;
        }
        if (j<ny/2){
          ky = j*ky0;
        }
        else{
          ky = (j-ny)*ky0;
        }
        if (k<nz/2){
          kz = k*kz0;
        }
        else{
          kz = (k-nz)*kz0;
        }
        k2 = kx*kx + ky*ky + kz*kz;
        e1 = exp(-(1-eta)*k2/4/xi/xi);
        Hx_tilde[0*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k] = e1*
        8*M_PI*(1+k2/4/xi/xi)/k2/k2*
        ((k2-kx*kx)*Hx_hat[0*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]+
        (-kx*ky)*Hx_hat[1*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]+
        (-kx*kz)*Hx_hat[2*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]);

        Hx_tilde[1*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k] = e1*
        8*M_PI*(1+k2/4/xi/xi)/k2/k2*
        ((-ky*kx)*Hx_hat[0*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]+
        (k2-ky*ky)*Hx_hat[1*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]+
        (-ky*kz)*Hx_hat[2*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]);

        Hx_tilde[2*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k] = e1*
        8*M_PI*(1+k2/4/xi/xi)/k2/k2*
        ((-kz*kx)*Hx_hat[0*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]+
        (-kz*ky)*Hx_hat[1*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]+
        (k2-kz*kz)*Hx_hat[2*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]);
      }
    }
  }
  IFFT3DGPU(Hx_tilde, Hx);
  Gaussian_Gridding_type2_OMP(Hx);
  free(Hx);
  free(Hx_hat);
  printf("k-pace part finished with %f(s)\n",(tt.toc()));
}

void kspace(){
  Timer tt;
  tt.tic();
  double *Hx;
  Hx = (double*)malloc(sizeof(double)*nx*ny*(nz)*3);
  Gaussian_Gridding_type1(Hx);
  complex<double> *Hx_hat;
  Hx_hat = (complex<double>*)malloc(sizeof(complex<double>)*nx*ny*(nz/2+1)*3);
  FFT3D(Hx, Hx_hat);
  complex<double> Hx_tilde[3*(nx)*(ny)*(nz/2+1)];
  double kx, ky, kz, k2, e1;
  double kx0=2*M_PI/Lx, ky0=2*M_PI/Ly, kz0=2*M_PI/Lz;
  for (int i = 0; i<nx; i++){
    for (int j = 0; j<ny; j++){
      for (int k = 0; k<nz/2+1; k++){
        if (i==0 && j==0 && k==0){
          Hx_tilde[0*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]={0,0};
          Hx_tilde[1*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]={0,0};
          Hx_tilde[2*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]={0,0};
          continue;
        }
        if (i<nx/2){
          kx = i*kx0;
        }
        else{
          kx = (i-nx)*kx0;
        }
        if (j<ny/2){
          ky = j*ky0;
        }
        else{
          ky = (j-ny)*ky0;
        }
        if (k<nz/2){
          kz = k*kz0;
        }
        else{
          kz = (k-nz)*kz0;
        }
        k2 = kx*kx + ky*ky + kz*kz;
        e1 = exp(-(1-eta)*k2/4/xi/xi);
        Hx_tilde[0*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k] = e1*
        8*M_PI*(1+k2/4/xi/xi)/k2/k2*
        ((k2-kx*kx)*Hx_hat[0*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]+
        (-kx*ky)*Hx_hat[1*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]+
        (-kx*kz)*Hx_hat[2*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]);

        Hx_tilde[1*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k] = e1*
        8*M_PI*(1+k2/4/xi/xi)/k2/k2*
        ((-ky*kx)*Hx_hat[0*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]+
        (k2-ky*ky)*Hx_hat[1*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]+
        (-ky*kz)*Hx_hat[2*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]);

        Hx_tilde[2*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k] = e1*
        8*M_PI*(1+k2/4/xi/xi)/k2/k2*
        ((-kz*kx)*Hx_hat[0*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]+
        (-kz*ky)*Hx_hat[1*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]+
        (k2-kz*kz)*Hx_hat[2*(nx)*(ny)*(nz/2+1)+i*(ny)*(nz/2+1)+j*(nz/2+1)+k]);
      }
    }
  }
  IFFT3D(Hx_tilde, Hx);
  Gaussian_Gridding_type2(Hx);
  free(Hx);
  free(Hx_hat);
  printf("k-pace part finished with %f(s)\n",(tt.toc()));
}

void selfcontribution(){
  Timer tt;
  tt.tic();
  double tmp = (4*xi)/sqrt(M_PI);
  for (long i = 0; i < np; i++) {
    vel[DIM*i+0] -= tmp*strength[DIM*i+0];
    vel[DIM*i+1] -= tmp*strength[DIM*i+1];
    vel[DIM*i+2] -= tmp*strength[DIM*i+2];
  }
  printf("Self Contribution part finished with %f(s)\n",(tt.toc()));

}

void writeout(){
  ofstream output(outputfile);
  output << "#Velocity obtained from Ewald summation" << endl;
  output << np << endl;
  for(long i = 0; i < np; i += 1){
    output << std::setprecision(17) << i << " " << vel[DIM*i+0] << " " <<
    vel[DIM*i+1] << " " <<
    vel[DIM*i+2]<< "\n" ;
  }
  output.close();
  output.clear();
  printf("Write files into %s\n",outputfile.c_str());
}

void freeall(){
  free(grid);
  free(particle);
  free(strength);
  free(vel);
  cout << "Dynamical variable destoyed!" << '\n';
}
